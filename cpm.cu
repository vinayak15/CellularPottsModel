#include "hip/hip_runtime.h"
/**********************************************************************
*
* cpm.cpp
*
* This file is part of VesselGen(3D)
*
* Copyright (C) 2016 -- Centre for Biomedical Image Analysis (CBIA)
* http://cbia.fi.muni.cz/
*
* VesselGen is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* VesselGen is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with VesselGen. If not, see <http://www.gnu.org/licenses/>.

* Authors: David Svoboda, 
*          Peter Kováč
*
* Description: A basic class defining the cellular Potts model in 3D
* extended by diffusion equations and further add-ons to enable
* realistic generation of cellular networks.
*
***********************************************************************/


// Here we consider boundary are always periodic
#ifndef BOUNDARY_PERIODIC
#define BOUNDARY_PERIODIC
#endif



#include <i3d/image3d.h>

#include "cpm.h"
#include "settings.h"
#include "initial.h"
#include "rnd_generators.h"
#include "edge_detection.h"
#include "cell.h"
#include "boundary.h"
#include <valarray>
#include <assert.h>
#include <i3d/regions.h>
#include <i3d/threshold.h>
#include "edge_detection.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include ""
#include<device_atomic_functions.h>
#include<hip/device_functions.h>
#include<Windows.h>

//-----------------------------------------------------------------------
CPM::CPM(IniHandler *cfg): params(cfg)
{
	 currentStep = 0;
	 sumOfPlannedSteps =
		(int) (*params)["cellular potts model"]["number of steps"];
	 int no_nbh =
		(int) (*params)["cellular potts model"]["neighbourhood"];
	 boltzmannTemp =
		(float) (*params)["cellular potts model"]["boltzmann temperature"];
	 sliceOrder =
		(int) (*params)["rendering"]["order of visualized slice"];

	 switch(no_nbh)
	 {
		  case 6	:
					nbh = i3d::nb3D_o6;
					break;
		  case 18	:
					nbh = i3d::nb3D_o18;
					break;
		  case 26	:
					nbh = i3d::nb3D_o26;
					break;
		  default	:
					DEBUG_REPORT("Neighbourhood set to default "
										"value (6 neighbours) ...");
					nbh = i3d::nb3D_o6;
	 }

	 // Establish ADHESION STRENGTHS
	 CreateTableOfAdhesions(adhesions, params);

	 // Read SHAPE CONSTRAINTS
	 targetVolume =
		(int) (*params)["cellular potts model"]["target volume"];
	 lambdaVolume =
		(int) (*params)["cellular potts model"]["lambda volume"];
	 lambdaSurface =
		(int) (*params)["cellular potts model"]["lambda surface"];

	 // Read relaxation time
	 relaxation = 
		(int) (*params)["cellular potts model"]["relaxation time"];
	 // Read CHEMICAL PARAMETERS
	 chemotaxis = (double) (*params)["pde"]["chemotaxis"];
	 secreteRate = (double) (*params)["pde"]["secr_rate"];
	 decayRate = (double) (*params)["pde"]["decay_rate"];
	 diffCoeff = (double) (*params)["pde"]["diff_coeff"];
	 diffTime = (double) (*params)["pde"]["dt"];
	 diffSpace = (double) (*params)["pde"]["dx"];
	 pdeIters = (int) (*params)["pde"]["pde iterations"];

	 edgeContainer = NULL;
}

//-----------------------------------------------------------------------
CPM::~CPM()
{
	// Deallocate cell structures
	for(unsigned int i = 0; i < cells.size(); i++)
	{
		delete cells[i];
	}

	if (edgeContainer)
		delete edgeContainer;
}

//-----------------------------------------------------------------------
void CPM::ShowCellVolumes() const
{

	 std::cout << "Expected volume of each cell:" << targetVolume << std::endl;
	 for (size_t i=0; i<cells.size(); i++)
	 {
		  std::cout << "(" << i << ":" << cells[i]->Volume() << ")";
	 }
	 std::cout << std::endl;

}

//-----------------------------------------------------------------------
void CPM::InitializePopulation(i3d::Image3d<i3d::GRAY16> &img)
{
	 DEBUG_REPORT("Creating the new population ...");

	 // Initialize an empty scene (3d image)
	 CreateEmptyScene(imgCellIDs, *params);

	 // Fill the scene with the initial number of cells (rough masks)
	 GenerateInitialPopulation(imgCellIDs, *params);

	 // Create cell structure for every cell_id
	 for (size_t i = 0; i <= imgCellIDs.GetMaxValue(); i++)
	 {
		 cells.push_back(new Cell());
	 }
	 MeasureVolumeOfCells();
	 DEBUG_REPORT("Creation of a new population completed.");

	 imgCellIDs.SaveImage("_newinitial.ics");

	 /**
	  * Initialize also the memory buffer for final image rendering.
	  * If we allocated it just before rendering, i.e. many times,
	  * we will slow down the whole simulation process.
	  */
	 imgRendered.CopyMetaData(imgCellIDs);

	 // Initialize chemoatractant's concentration plane
	 imgConcentration.CopyMetaData(imgCellIDs);
	 imgAltConcentration.CopyMetaData(imgCellIDs);
	 img = imgCellIDs;

}

//-----------------------------------------------------------------------
void CPM::ImposeInitialPopulation(i3d::Image3d<i3d::GRAY16> &img)
{
	 imgCellIDs = img;

	 // Create cell structure for every cell_id
	 for (size_t i = 0; i <= imgCellIDs.GetMaxValue(); i++)
	 {
		 cells.push_back(new Cell());
	 }
	 MeasureVolumeOfCells();

	 /**
	  * Initialize also the memory buffer for final image rendering.
	  * If we allocated it just before rendering, i.e. many times,
	  * we will slow down the whole simulation process.
	  */
	 imgRendered.CopyMetaData(imgCellIDs);

	 // Initialize chemoatractant's concentration plane
	 imgConcentration.CopyMetaData(imgCellIDs);
	 imgAltConcentration.CopyMetaData(imgCellIDs);

	 DEBUG_REPORT("Initial cell population read from file you provided.");
}

//-----------------------------------------------------------------------
void CPM::MeasureVolumeOfCells()
{
	// Set volumes of cells
	for (size_t i = 0; i < imgCellIDs.GetImageSize(); i++)									//Error in code
	{
		if (GetObjectType(imgCellIDs.GetVoxel(i)) == TypeCell) 
		{
			cells[imgCellIDs.GetVoxel(i)]->IncrementVolume();
		}
	}
}

//-----------------------------------------------------------------------
void CPM::PrecomputeEdges()
{
	 edgeContainer = new Edges(this->imgCellIDs, this->nbh);
}

//-----------------------------------------------------------------------
void CPM::DoNextStep(float * imgCellIDs, float *imgConcentration)
{
	DEBUG_REPORT("Current step is: " << currentStep);

	size_t rand;

	for (size_t i = 0; i < edgeContainer->size(); i++)
	{
		// 1. Select RANDOM 'source' voxel from edgeSet
		rand = GetRandomUniform(0, edgeContainer->size() - 1);
		//std::set<i3d::Vector3d<int> >::const_iterator it(edgeSet.begin());

		/// OBSOLETE (too slow)
		// 'advance' the iterator RAND times (linear complexity)
		//std::advance(it, rand);

		// random selection of edge voxel with O(1) complexity
		size_t j = edgeContainer->GetIndex(rand);
		i3d::Vector3d<int> sourceCoor = i3d::Vector3d<size_t>(this->imgCellIDs.GetX(j), this->imgCellIDs.GetY(j), this->imgCellIDs.GetZ(j));

		// 2. Load the value of the source voxel
		i3d::GRAY16 sourceValue = imgCellIDs[this->imgCellIDs.GetIndex(sourceCoor)];

		if (sourceValue == ID_ECM)
			DEBUG_REPORT("CHYBA!");

		// 3. Select RANDOM neighbour of selected voxel
		//
		// If we stay in the relaxation time, do not leave the selection
		// process to be random. Prefer the bottom neighbours.
		int selection = GetRandomUniform(1, nbh.size() - 1);

		if ((int)currentStep <= relaxation)
		{
			i3d::Vector3d<int> pos = nbh.offset[selection];

			// The new z-position should be deeper or the same. If not, repeat the 
			// selection process.
			while (pos.z > -1)
			{
				selection = GetRandomUniform(1, nbh.size() - 1);
				pos = nbh.offset[selection];
			}
		}

		i3d::Vector3d<int> targetCoor = sourceCoor + nbh.offset[selection];

		// 4. Check boundary condition
		if (!ValidateCoords(targetCoor, this->imgCellIDs.GetSize()))
		{
			continue;
		}

		// 5. Load the value of the target voxel
		i3d::GRAY16 targetValue = imgCellIDs[this->imgCellIDs.GetIndex(targetCoor)];

		// The value of source voxel is always some cell ID! We need to check
		// the value of the target voxel. It cannot be ID_ECM (it is a solid
		// material and the cells cannot penetrate it). Additionally, it cannot
		// bear the same value as the source voxel does. If so, no change
		// happens.
		if ((targetValue != ID_ECM) && (sourceValue != targetValue))
		{
			if (targetValue == ID_ECM)
				DEBUG_REPORT("Divne targetValue");
			if (sourceValue == ID_ECM)
				DEBUG_REPORT("Divne sourceValue");

			assert((targetValue != ID_ECM) && (sourceValue != ID_ECM));
			double deltaH = 0;

			// propose the spin flip and compute the difference of Hamiltonian
			deltaH = ComputeDeltaH(sourceCoor, targetCoor, imgCellIDs, imgConcentration);

			// is the change accepted?
			if (ProbabilityToSpin(deltaH))
			{
				// do the change
				PerformSpin(sourceCoor, targetCoor, imgCellIDs);

				// update the list of edges
				edgeContainer->Update(this->imgCellIDs, nbh, sourceCoor, imgCellIDs);
			}
		}
	}

	// keep this incrementation at the end of this method!
	currentStep++;
}

//-----------------------------------------------------------------------
bool CPM::ProbabilityToSpin(double deltaH)
{
	double prob;
	//if deltaH = 0 => spin will be performed with 100 percent probability
	if (deltaH <= 0)
	{
		return true;
	}
	else
	{
		//computing probability of spin based on deltaH
		//larger deltaH means larger probability
		prob = exp(-(deltaH / boltzmannTemp));
		return GetRandomUniform(0.0f, 1.0f) < prob;
	}
}

//-----------------------------------------------------------------------
void CPM::PerformSpin(i3d::Vector3d<int> source, i3d::Vector3d<int> target, float * imgCellIDs)
{
	i3d::GRAY16 sourceID = imgCellIDs[this->imgCellIDs.GetIndex(source)];
	i3d::GRAY16 targetID = imgCellIDs[this->imgCellIDs.GetIndex(target)];

	cells[sourceID]->DecrementVolume();
	cells[targetID]->IncrementVolume();

	imgCellIDs[this->imgCellIDs.GetIndex(source)] = imgCellIDs[this->imgCellIDs.GetIndex(target)];
}

//-----------------------------------------------------------------------
int CPM::LocalSurfaceAfterChange(i3d::Vector3d<int> changingCoor,
	int cellID, int newID, float *imgCellIDs)
{
	//	localSurface -> number of voxels with cellID that create surface 
	//					after performing change
	//				 -> these voxels belong to neighbourhood of changingCoor 
	//					(changingCoor included)
	int localSurface = 0;
	std::vector<const float *> winNbh;
	//	iterate through all neighbour voxels of changingCoor voxel
	for (size_t i = 0; i < nbh.size(); i++) {
		i3d::Vector3d<int> nbhCoor(changingCoor + nbh.offset[i]);
		// handle boundary conditions
		if (!ValidateCoords(nbhCoor, this->imgCellIDs.GetSize()))
		{
			continue;
		}
		//	skip current voxels with another ID than cellID
		if (imgCellIDs[this->imgCellIDs.GetIndex(nbhCoor)] != cellID)
		{
			continue;
		}
		// This condition is here because of OPTIMIZATION.
		// At this point we know that "current voxel" has id=cellID and
		// is neighbour of voxel with coordinates=changeCoor.
		// ChangeCoor voxel has id=newID. If newID is different from cellID, 
		// we can say "current voxel"(the one with id=cellID) belongs to SURFACE
		if (imgCellIDs[this->imgCellIDs.GetIndex(nbhCoor)] != newID)
		{
			localSurface++;
			continue;
		}
		// If program gets to this place in code, 
		// it means newID = cellID => current voxel - cellID
		//							  changeCoor voxel - cellID
		// We then have to check whole neighbourhood of current voxel and
		// decide if it creates SURFACE
		//take neighbourhood of site

		i3d::Neighbourhood rnb;
		size_t tmp = GetNbh(this->imgCellIDs, nbhCoor.x, nbhCoor.y, nbhCoor.z, nbh, rnb);
		winNbh.resize(nbh.size());
		i3d::VectContainer::const_iterator off;
		int q = 0;
		for (off = nbh.offset.begin(); off != nbh.offset.end(); ++off)
		{
			winNbh[q++] = &imgCellIDs[nbhCoor.x + off->x + (nbhCoor.y + off->y)*this->imgCellIDs.GetSizeX() + (nbhCoor.z + off->z)*this->imgCellIDs.GetSizeX() * this->imgCellIDs.GetSizeY()];
		}


		for (size_t j = 1; j < winNbh.size(); j++)
		{
			// check if site is on surface
			if (imgCellIDs[this->imgCellIDs.GetIndex(nbhCoor)] != *winNbh[j])
			{
				localSurface++;
				break;
			}
		}
	}
	return localSurface;
}

//-----------------------------------------------------------------------
double CPM::ComputeDeltaH(i3d::Vector3d<int> source, i3d::Vector3d<int> target, float *imgCellIDs, float * imgConcentration)
{
	double H_before = 0.0;
	double H_after = 0.0;

	i3d::GRAY16 sourceID = imgCellIDs[this->imgCellIDs.GetIndex(source)];
	i3d::GRAY16 targetID = imgCellIDs[this->imgCellIDs.GetIndex(target)];
	i3d::GRAY8 nbhID;

	// *** H_adhesion ***
	//
	// In the following loop we need to inspect, how the possible change
	// affects the relations ship of all the neighbouring voxels.
	//
	for (size_t i = 1; i < nbh.size(); i++)
	{
		i3d::Vector3d<int> nbhCoor = source + nbh.offset[i];
		// handle boundary condition
		if (!ValidateCoords(nbhCoor, this->imgCellIDs.GetSize()))
		{
			continue;
		}
		nbhID = imgCellIDs[this->imgCellIDs.GetIndex(nbhCoor)];

		H_after += adhesions[GetObjectType(targetID)][GetObjectType(nbhID)];
		H_before += adhesions[GetObjectType(sourceID)][GetObjectType(nbhID)];
	}

	// *** H_shape ***
	//
	// VOLUME CONSTRAINT
	//
	// Here, we inspect, how the spin spin affects the volume of the object
	// which the source voxel belongs to.
	//
	if (targetID == ID_MEDIUM)
	{
		//situation -> source CELL, target MEDIUM
		//if change happens, volume of cell will decrease by one
		H_before += lambdaVolume *
			SQRd((cells[sourceID]->Volume() - TargetVolume()));

		H_after += lambdaVolume *
			SQRd((cells[sourceID]->Volume() - 1 - TargetVolume()));

	}
	else if (sourceID == ID_MEDIUM)
	{
		//situation -> source MEDIUM, target CELL
		//if change happens, volume of cell will increase by one
		H_before += lambdaVolume *
			SQRd((cells[targetID]->Volume() - TargetVolume()));

		H_after += lambdaVolume *
			SQRd((cells[targetID]->Volume() + 1 - TargetVolume()));
	}
	else
	{
		//situation -> source CELL1, target CELL2
		//if change happens, volume of cell1 will decrease by one
		//					 volume of cell2 will increase by one
		H_before += lambdaVolume * (
			SQRd((cells[sourceID]->Volume() - TargetVolume())) +
			SQRd((cells[targetID]->Volume() - TargetVolume())));
		H_after += lambdaVolume * (
			SQRd((cells[sourceID]->Volume() - 1 - TargetVolume())) +
			SQRd((cells[targetID]->Volume() + 1 - TargetVolume())));

	}

	//
	// SURFACE CONSTRAINT
	// 
	if (targetID == ID_MEDIUM)
	{
		// get local surface of cell with id=sourceID before performing spin 
		// change on coordinate "source"
		int deltaSurBefore = LocalSurfaceAfterChange(source, sourceID, sourceID, imgCellIDs);
		// get local surface of cell with id=sourceID after performing spin 
		// change on coordinate "source"
		int deltaSurAfter = LocalSurfaceAfterChange(source, sourceID, targetID, imgCellIDs);
		//situation -> source CELL, target MEDIUM
		H_before += lambdaSurface * deltaSurBefore;
		H_after += lambdaSurface * deltaSurAfter;
	}
	else if (sourceID == ID_MEDIUM)
	{
		// get local surface of cell with id=targetID before performing spin 
		// change on coordinate "source"	
		int deltaSurBefore = LocalSurfaceAfterChange(source, targetID, sourceID, imgCellIDs);
		// get local surface of cell with id=targetID after performing spin 
		// change on coordinate "source"
		int deltaSurAfter = LocalSurfaceAfterChange(source, targetID, targetID, imgCellIDs);
		//situation -> source CELL, target MEDIUM
		H_before += lambdaSurface * deltaSurBefore;
		H_after += lambdaSurface * deltaSurAfter;
	}
	else
	{
		// get local surface of cell with id=sourceID on coordinate "source" and
		// local surface of cell with id=targetID on coordinate "target" BEFORE
		// performing spin change
		int deltaSurBefore = LocalSurfaceAfterChange(source, sourceID,
			sourceID, imgCellIDs) + LocalSurfaceAfterChange(target, targetID, targetID, imgCellIDs);
		// get local surface of cell with id=sourceID on coordinate "source" and
		// local surface of cell with id=targetID on coordinate "target" AFTER
		// performing spin change
		int deltaSurAfter = LocalSurfaceAfterChange(source, sourceID, targetID, imgCellIDs);
		// to get local surface of cell with id=targetID after spin change, 
		// we need to change voxel value on coordinate "source" to targetID
		// and then after applying function LocalSurfaceAfterChange return it
		// back again
		imgCellIDs[this->imgCellIDs.GetIndex(source)] = targetID;
		deltaSurAfter += LocalSurfaceAfterChange(target, targetID, targetID, imgCellIDs);
		imgCellIDs[this->imgCellIDs.GetIndex(source)] = sourceID;
		//situation -> source CELL1, target CELL2
		H_before += lambdaSurface * deltaSurBefore;
		H_after += lambdaSurface * deltaSurAfter;
	}

	/// *** H_chemical ***
	double DDH = 0;

	DDH = (chemotaxis * (imgConcentration[this->imgCellIDs.GetIndex(source)] -
		imgConcentration[this->imgCellIDs.GetIndex(target)]));

	return (H_after - H_before) - DDH;
}

//-----------------------------------------------------------------------

//-----------------------------------------------------------------------
void CPM::Secrete() 
{
	const double increase = secreteRate * diffTime;
	const double decay = (1.0f - decayRate);
	const size_t sz = imgCellIDs.GetImageSize();
	const i3d::GRAY16 *currIDPtr = imgCellIDs.GetFirstVoxelAddr();
	float *currConcPtr = imgConcentration.GetFirstVoxelAddr();

	for (size_t i=0; i<sz; i++)
	{
		 // Only the cells secrete chemoattractant, medium and ECM does not
		 if ((*(currIDPtr) != ID_MEDIUM) && (*(currIDPtr) != ID_ECM))
		 {
			  *(currConcPtr) += increase;
		 }
		 else // Outside the cells, chemoattractant decays
		 {
			  *(currConcPtr) *= decay;
		 }

		 currIDPtr++;
		 currConcPtr++;
	}

}

//---------------------------------------------------------------------------------------------


__global__ void CudaSecrete(float *imgCellIDs, float *imgConcentration, float secreteRate, float diffTime, float decayRate)      //Decalred global so that it can access by any class
{
	size_t index = threadIdx.x + blockIdx.x*blockDim.x;																//threadIdx.x is index of thread in x direction and blockIdx.x is index of block in X direction  and blockDim.x is dimension of block in x direction
	const double increase = secreteRate*diffTime;							
	const double decay=(1.0f - decayRate);
	if (imgCellIDs[index] != ID_MEDIUM && imgCellIDs[index] != ID_ECM)			
		imgConcentration[index] += increase;
	else
		imgConcentration[index] *= decay;
}

//---------------------------------------------------------------------------------------------
__global__ void CudaDiffuse(float *imgConcentration, float *imgAltConcentration, int width, int height, int depth, int sliceSize, double diffConstant)
{
	size_t index = threadIdx.x + blockIdx.x*blockDim.x;
	size_t x = index%width;
	size_t y = (index / width) % height;
	size_t z = ((index / width) / height);
	size_t left, right, above, below, front, behind;
	float sum;
	left = index - 1;																//Setting left,right, above, below,front, behind index for each and every thread.
	right = index + 1;
	above = index - width;																
	below = index + width;
	front = index - sliceSize;
	behind = index + sliceSize;

#ifdef BOUNDARY_PERIODIC															//Treating each and every boundary condition seprately
	if ( x == 0 &&  y == (height - 1) &&  z<(depth - 1) &&  z>0) {		//here block Idx.x is index fof image in x direction  
		left = index + (width - 1);
		below = index - width*(height - 1);															// y is index of image in y direction
		// diffuse chemoattractant in every voxel (except for boundary ones)						// z is index of image in z direction
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( x == (width - 1) &&  (y == height - 1) &&  z<depth - 1 &&  z>0) {
		right = index - (width - 1);
		below = index - width*(height - 1);

		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( x == (width - 1) &&  y == 0 &&  z<depth - 1 &&  z>0) {
		right = index - (width - 1);
		above = index + width*(height - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( x == 0 &&  y == 0 &&  z<(depth - 1) &&  z>0) {
		left = index + (width - 1);
		above = index + width*(height - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( x == (width - 1) &&  z<(depth - 1) &&  z>0) {
		right = index - (width - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( x == 0 &&  z<depth - 1 &&  z>0) {
		left = index + (width - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( y == height - 1 &&  z<depth - 1 &&  z>0) {
		below = index - width*(height - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( y == 0 &&  z<depth - 1 &&  z>0) {
		above = index + width*(height - 1);
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	else if ( (x >= 1) && ( y >= 1) && ( x<(width - 1)) &&  (y<(height - 1)) &&  (z<(depth - 1)) &&  (z>0) )
	{
		// diffuse chemoattractant in every voxel (except for boundary ones)
		sum = (-6.0f) * (imgConcentration[index]);
		sum += (imgConcentration[left]) + (imgConcentration[right]) + (imgConcentration[above]) + (imgConcentration[below]) + (imgConcentration[front]) + (imgConcentration[behind]);

		// store the result
		imgAltConcentration[index] = imgConcentration[index] + sum * diffConstant;
	}
	#endif
												
	imgConcentration[index] = imgAltConcentration[index];
	
}


//-----------------------------------------------------------------------
void CPM::Render(float *imgCellIDs)
{
	 DEBUG_REPORT("Creating the output image");
	 for (size_t i = 0; i < this->imgCellIDs.GetImageSize(); i++)
		 this->imgCellIDs.SetVoxel(i, imgCellIDs[i]);
	 const char *colors = (*params)["rendering"]["true colors"];

	 if (strcmp(colors, "true") == 0)
	 {
		GrayToRGB(this->imgCellIDs, this->imgCellIDs, this->imgCellIDs, imgRendered);
	 }
	 else if (strcmp(colors, "false") == 0)
	 {
		  i3d::RGB16 rgbValue;

		  for (size_t i=0; i<imgRendered.GetImageSize(); i++)
		  {
				i3d::GRAY16 cellID = this->imgCellIDs.GetVoxel(i);

				if (cellID == ID_MEDIUM)
				{
					 rgbValue.red = rgbValue.green = rgbValue.blue = 0;
				}
				else if (cellID == ID_ECM)
				{
					 rgbValue.red = rgbValue.green = rgbValue.blue = 128;
				}
				else // the places where the cells are located
				{
					 // yellow color
					 rgbValue.red = 255;
					 rgbValue.green = 255;
					 rgbValue.blue = 0;
				}

				imgRendered.SetVoxel(i, rgbValue);
		  }
	 }
	 else
	 {
		  throw ERROR_REPORT("Unknown key value. Expected 'true'/'false'.");
	 }


	 DEBUG_REPORT("Image completed");
}

void CPM::Render()
{
	DEBUG_REPORT("Creating the output image");

	const char *colors = (*params)["rendering"]["true colors"];

	if (strcmp(colors, "true") == 0)
	{
		GrayToRGB(imgCellIDs, imgCellIDs, imgCellIDs, imgRendered);
	}
	else if (strcmp(colors, "false") == 0)
	{
		i3d::RGB16 rgbValue;

		for (size_t i = 0; i<imgRendered.GetImageSize(); i++)
		{
			i3d::GRAY16 cellID = imgCellIDs.GetVoxel(i);

			if (cellID == ID_MEDIUM)
			{
				rgbValue.red = rgbValue.green = rgbValue.blue = 0;
			}
			else if (cellID == ID_ECM)
			{
				rgbValue.red = rgbValue.green = rgbValue.blue = 128;
			}
			else // the places where the cells are located
			{
				// yellow color
				rgbValue.red = 255;
				rgbValue.green = 255;
				rgbValue.blue = 0;
			}

			imgRendered.SetVoxel(i, rgbValue);
		}
	}
	else
	{
		throw ERROR_REPORT("Unknown key value. Expected 'true'/'false'.");
	}


	DEBUG_REPORT("Image completed");
}


//-----------------------------------------------------------------------
void CPM::StoreToFile() const
{
	 const char *agreement = (*params)["rendering"]["store data to disk"];

	 if (strcmp(agreement,"true") == 0)
	 {
		  char fname[MAX_STRLEN];
		  sprintf(fname, "img_%.4lu.ics", currentStep);

		  DEBUG_REPORT("Saving the image file: " << fname);
		  imgRendered.SaveImage(fname);
		  DEBUG_REPORT("File saved successfully.");

		  /*
		  // TODO: smazat po odladeni
		  // begin - koncentrace (separe) 
		  char fname2[MAX_STRLEN];
		  sprintf(fname2, "signals_%.4lu.ics", currentStep);
		  DEBUG_REPORT("Saving the image file: " << fname2);
		  imgConcentration.SaveImage(fname2);
		  DEBUG_REPORT("File saved successfully.");
		  // end - koncentrace
		  */
	 }
	 else if (strcmp(agreement,"false") == 0)
	 {
		  // do nothing
	 }
	 else
	 {
		  throw ERROR_REPORT("Unknown key value. Expected 'true'/'false'");
	 }
}
//-----------------------------------------------------------------------
void CPM::GetImage(float *img) {
	for (size_t i = 0; i < imgCellIDs.GetImageSize(); i++)
		img[i] = imgCellIDs.GetVoxel(i);
}

//-----------------------------------------------------------------------

void CPM::SetImage(float *imgCon) {
	for (size_t i = 0; i < imgCellIDs.GetImageSize(); i++) {
		imgConcentration.SetVoxel(i, imgCon[i]);
	}

}

//-----------------------------------------------------------------------
ObjectType GetObjectType(int objectID)
{
	 if (objectID < 0)
	 {
		  ERROR_REPORT("Invalid ID value!");
	 }

	 switch (objectID)
	 {
		  case ID_MEDIUM: return TypeMedium;
		  case ID_ECM: return TypeECM;
		  default: {};
	 }

	 return TypeCell;
}


//-----------------------------------------------------------------------
