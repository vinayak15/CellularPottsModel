#include "hip/hip_runtime.h"
﻿/**********************************************************************
* 
* main.cpp
*
* This file is part of VesselGen(3D)
* 
* Copyright (C) 2016 -- Centre for Biomedical Image Analysis (CBIA)
* http://cbia.fi.muni.cz/
* 
* VesselGen is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
* 
* VesselGen is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
* 
* You should have received a copy of the GNU General Public License
* along with VesselGen. If not, see <http://www.gnu.org/licenses/>.
* 
* Author: David Svoboda
* 
* Description: Main body of the application (mostly the interface).
*
***********************************************************************/

//Here we define vessel3d_debug if not define
// Here defne debug if not define

#ifndef VESSEL3D_DEBUG
#define VESSEL3D_DEBUG
#endif
#ifndef DEBUG
#define DEBUG
#endif



#include <iostream>
#include "XGetopt.h"
#include<time.h>
#ifdef QT_SUPPORT
	#include <QtWidgets/QApplication>
	#include <QtWidgets/QLabel>
	#include <QtGui/QPixmap>
	#include "qtdisplay.h"
#endif

#include "ini/iniparser.h"
#include "settings.h"
#include "cpm.h"
#include "edge_detection.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include ""
#include<device_atomic_functions.h>
#include<hip/device_functions.h>


using namespace std;

//----------------------------------------------------------------------------------
// The function describing the use of this console application
// it is called whenever the program is called without parameters
// or with bad combination of parameters
//----------------------------------------------------------------------------------
void usage(const char *name)
{
	 cerr << "APPLICATION NAME" << endl <<
				"\t" << name << " ... 3D dynamic vessel generator  " <<
				endl << endl;
	 cerr << "SYNTAX" << endl << "\t" << name << " <options> " << endl << endl;
	cerr << "OPTIONS" << endl <<
				"\t-c <filename> ... ini file containing the description\n" <<
				"\t                  of the simulation (mandatory option)\n" <<
				"\t-p <filename> ... image with pregenerated cell population\n" <<
				"\t                  (must be compatible with the ini file)\n" <<
				"\t-h            ... this help" << endl << endl;

	 cerr << "SOME EXAMPLES:" << endl << endl;

	 cerr << "** " << "Basic generation without use of any pregenerated data:" << 
				endl <<
				"\t" << name << " -c vessel.ini" << endl << endl;
	 cerr << "** " << "Generation based on already pregenerated cell population: " <<
				endl << 
				"\t" << name << " -c config.ini -p data/cells_25.ics" <<
				endl << endl;
}

//----------------------------------------------------------------------------------
// Entry point of the program, the main() function.
//----------------------------------------------------------------------------------
int main(int argc, char *argv[])
{
	 try {

	 // string variables with filenames
	 std::string 
				iniFilename(""),
				pregeneratedImageFilename("");

	 
	 // Read the command line input
	 if (argc < 2)
	 {
		  usage(argv[0]);
		  exit(-1);
	 }

	 // variable that controls the 'getopt' function
	 int c;

	 // read the command line options
	 while ((c = getopt(argc, argv, "c:p:h")) != -1)
	 {
			switch (c)
			{
					 case 'c': iniFilename = optarg;
								  break;
					 case 'p': pregeneratedImageFilename = optarg;
								  break;
					 case 'h': // user asks for help
					 default: usage(argv[0]);
								 exit(-1);
			}
	 }

	 if (iniFilename.empty())
	 {
		  cerr << "\nERROR: You missed to enter the configuration file! \n\n";
		  usage(argv[0]);
		  exit(-1);
	 }


	 DEBUG_REPORT("supplied configuration file: " << iniFilename.c_str());
	 IniHandler configuration(iniFilename.c_str());

	 // Initialize random seed generator (either manually or automatically)
	 unsigned int RandSeed;

	 if (configuration["cellular potts model"].present("random seed"))
	 {
		  RandSeed = (int) configuration["cellular potts model"]["random seed"];
	 }
	 else
	 {
		  RandSeed = time(NULL);
	 }

	DEBUG_REPORT("stdlib generator seeded with " << RandSeed); 
	 srand(RandSeed);

	 // Create the basic model
	 i3d::Image3d<i3d::GRAY16> img;
	 CPM model(&configuration);

	 // Set up the initial cell population
	 if (pregeneratedImageFilename.empty())
	 { 
		  model.InitializePopulation(img);
	 }
	 else
	 {
		  img= *(new i3d::Image3d<i3d::GRAY16>(pregeneratedImageFilename.c_str()));						//taking content of new object into img
		  model.ImposeInitialPopulation(img);
	 } 

	 //  Detect edges between cell population and medium
	 model.PrecomputeEdges();
/*	 EdgeDetection(model.ShowPopulation(), 
						model.GetNeighbourhood(), 
						model.GetEdgeSet());*/

	 model.Render();
	 model.StoreToFile();

	 float secreteRate = (float) configuration["pde"]["secr_rate"];
	 float decayRate = (float) configuration["pde"]["decay_rate"];
	 float diffCoeff = (float) configuration["pde"]["diff_coeff"];
	 float  diffTime = (float)configuration["pde"]["dt"];
	 float diffSpace = (float) configuration["pde"]["dx"];
	 double diffConst = (diffCoeff*diffTime) / (diffSpace*diffSpace);

	 
	 // The main application loop
#ifdef QT_SUPPORT
	 QApplication app(argc, argv);
	 MainWidget w(&model);
	 app.connect(&w, SIGNAL(SimulationCompleted(void)), SLOT(quit(void)) );
	app.exec();
#else
//----------------------------------------------------------------------------
	//Vinayak Changes
	long long  t1, t2, t3, t4, t5,t6,t7, t8;
	double secreteTime =0.0 , diffuseTime=0.0 , loopTime=0.0 , donextTime=0.0;
	 float *imgCellIDs, *cudaimgCellIDs, *cudaimgConcentration,									//Declaring Cuda Kernel Variable
		 *cudaimgAltConcentration , *imgConcentration;
	 imgCellIDs = new float[img.GetImageSize()];									
	 imgConcentration = new float[img.GetImageSize()];
	 hipMalloc((void **)&cudaimgCellIDs, sizeof(float)*img.GetImageSize());					//Allocating Memory for Cuda Kernel Variable  
	 hipMalloc((void **)&cudaimgAltConcentration, sizeof(float)*img.GetImageSize());			//Allocating Memory for Cuda Kernel Variable
	 hipMalloc((void **)&cudaimgConcentration, sizeof(float)*img.GetImageSize());				//Allocating Memory for Cuda Kernel Variable
	
	 model.GetImage(imgCellIDs);														

	 hipMemcpy(cudaimgCellIDs, imgCellIDs, sizeof(float)*img.GetImageSize(), hipMemcpyHostToDevice);	//Copying CPU Memory to Kernel Memory(from CPU to GPU)

	 hipMemset(cudaimgConcentration, 0.0, sizeof(float)*img.GetImageSize());							//Intializing imgConcentration to 0 in GPU 
	 hipMemset(cudaimgAltConcentration, 0.0, sizeof(float)*img.GetImageSize());						//Intializing imgConcentration to 0 in GPU
	 int renderingPeriod = configuration["rendering"]["period"];


    t1=clock();
	 for (size_t it = 0; it < model.GetOverallDuration(); it++) 
	 {
            t2=clock();
			for (int r = 0; r < model.PdeIterations(); r++) 
			{
		        t3=clock();
				CudaSecrete << <img.GetImageSize() / 1024, 1024 >> > (cudaimgCellIDs,					//Calling Secrete Function in GPU with number of threads Per block to be 1024 as maximum capacity for this system . And passing number of blocks to be img.GetImageSize()/1024 and total number of threads are to be (img.GetImageSize()/1024)*1024. 
					cudaimgConcentration, secreteRate, diffTime, decayRate);



				t4=clock();
				CudaDiffuse << <img.GetImageSize() / 1024, 1024 >> > (cudaimgConcentration, cudaimgAltConcentration,					//Calling Diffuse function in GPU with number of threads per block to be 1 and nubmer of blocks toequal to image size distrubuted in X , Y ,Z direction according to image. 
					img.GetSizeX(), img.GetSizeY(), img.GetSizeZ(), img.GetSizeX()*img.GetSizeY(), diffConst);
				t5=clock();	
				
				secreteTime+= (double)(t4-t3)/double(CLOCKS_PER_SEC)*1000.0;
				diffuseTime+= (double)(t5-t4)/double(CLOCKS_PER_SEC)*1000.0;
			}

			//model.ShowCellVolumes();
			t6=clock();
			loopTime+=(double)(t5-t2)/double(CLOCKS_PER_SEC)*1000.0;
			
			hipMemcpy(imgConcentration, cudaimgConcentration, sizeof(float)*img.GetImageSize(), hipMemcpyDeviceToHost);		//Calling Cuda Memcpy to copy contents of CUda kernel memory to CPU memory .In PDe iterations only cudaimgConcentration was chaged so to copy contents of cudaimgConcentration to imgConcentration.

			model.DoNextStep(imgCellIDs, imgConcentration);

			
			hipMemcpy(cudaimgCellIDs, imgCellIDs, sizeof(float)*img.GetImageSize(), hipMemcpyHostToDevice);//Calling CUda Memcpy to copy contents of CPU memory to cuda Kernel Memory .In DonextStep only imgCellIDs was chaged so to copy contents of imgCellIDs to cudaimgCellIDs.
			t7=clock();
			if (it%renderingPeriod == 0) {
				model.Render(imgCellIDs);
				model.StoreToFile();

			}
            donextTime+= (double)(t7-t6)/double(CLOCKS_PER_SEC)*1000.0;
	 }
	 t8=clock();
     cout << "\n Mean GPU Time Taken By whole pRocess = " << ((t8 - t1)/double(CLOCKS_PER_SEC)*1000)/model.GetOverallDuration() << "\n";
	 cout << "\n Mean GPU TIme Taken By loop = " << loopTime / (model.GetOverallDuration()) << "\n";
	 cout << "\n Mean GPU TIme Taken By DoNextStep = " << donextTime / (model.GetOverallDuration()) << "\n";
	 cout << "\n Mean GPU TIme for secrete function = " << secretTime / (model.GetOverallDuration()*model.PdeIterations()) << "\n";
	 cout << "\n Mean GPU Time for Diffuse Funciton = " << diffuseTime / (model.GetOverallDuration()*model.PdeIterations()) << "\n";




	 delete[] imgCellIDs;
	 delete[] imgConcentration;
	 hipFree(cudaimgCellIDs);
	 hipFree(cudaimgAltConcentration);
	 hipFree(cudaimgConcentration);
	 
//End of chagnes
#endif

	 } catch (std::string &e)
		  {
				cout << e << endl;
		  }
		  catch (i3d::IOException& e)
		  {
				cout << e << endl;
		  }
		  catch (i3d::InternalException& e)
		  {
				cout << e << endl;
		  }
		  catch (std::bad_alloc&)
		  {
				cout << "Not enough memory." << endl;
		  }
		  catch (...)
		  {
				cout << "System exception (2)." << endl;
		  } 

	 return 0;
}
